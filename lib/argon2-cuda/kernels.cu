#include "hip/hip_runtime.h"
/* For IDE: */
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "kernels.h"
#include "cudaexception.h"

#include <stdexcept>
#ifndef NDEBUG
#include <iostream>
#endif

#define ARGON2_D 0
#define ARGON2_I 1

#define ARGON2_VERSION_10 0x10
#define ARGON2_VERSION_13 0x13

#define ARGON2_BLOCK_SIZE 1024
#define ARGON2_QWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 8)
#define ARGON2_SYNC_POINTS 4

#define THREADS_PER_LANE 32
#define QWORDS_PER_THREAD (ARGON2_QWORDS_IN_BLOCK / 32)

namespace argon2 {
namespace cuda {

using namespace std;

__device__ uint64_t u64_build(uint32_t hi, uint32_t lo)
{
    return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint32_t u64_lo(uint64_t x)
{
    return (uint32_t)x;
}

__device__ uint32_t u64_hi(uint64_t x)
{
    return (uint32_t)(x >> 32);
}

struct block_g {
    uint64_t data[ARGON2_QWORDS_IN_BLOCK];
};

struct block_l {
    uint32_t lo[ARGON2_QWORDS_IN_BLOCK];
    uint32_t hi[ARGON2_QWORDS_IN_BLOCK];
};

__device__ void move_block(struct block_l *dst, const struct block_l *src,
                           uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = i * THREADS_PER_LANE +
                (thread & 0x10) + ((thread + i * 4) & 0xf);
        dst->lo[pos_l] = src->lo[pos_l];
        dst->hi[pos_l] = src->hi[pos_l];
    }
}

__device__ void xor_block(struct block_l *dst, const struct block_l *src,
                          uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = i * THREADS_PER_LANE +
                (thread & 0x10) + ((thread + i * 4) & 0xf);
        dst->lo[pos_l] ^= src->lo[pos_l];
        dst->hi[pos_l] ^= src->hi[pos_l];
    }
}

__device__ void load_block(struct block_l *dst, const struct block_g *src,
                           uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = (thread & 0x10) + ((thread + i * 4) & 0xf);
        uint64_t in = src->data[i * THREADS_PER_LANE + thread];
        dst->lo[i * THREADS_PER_LANE + pos_l] = u64_lo(in);
        dst->hi[i * THREADS_PER_LANE + pos_l] = u64_hi(in);
    }
}

__device__ void load_block_xor(struct block_l *dst, const struct block_g *src,
                               uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = (thread & 0x10) + ((thread + i * 4) & 0xf);
        uint64_t in = src->data[i * THREADS_PER_LANE + thread];
        dst->lo[i * THREADS_PER_LANE + pos_l] ^= u64_lo(in);
        dst->hi[i * THREADS_PER_LANE + pos_l] ^= u64_hi(in);
    }
}

__device__ void store_block(struct block_g *dst, const struct block_l *src,
                            uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = (thread & 0x10) + ((thread + i * 4) & 0xf);
        uint64_t out = u64_build(src->hi[i * THREADS_PER_LANE + pos_l],
                                 src->lo[i * THREADS_PER_LANE + pos_l]);
        dst->data[i * THREADS_PER_LANE + thread] = out;
    }
}

__device__ uint64_t rotr64(uint64_t x, uint32_t n)
{
    return (x >> n) | (x << (64 - n));
}

__device__ uint64_t f(uint64_t x, uint64_t y)
{
    uint32_t xlo = u64_lo(x);
    uint32_t ylo = u64_lo(y);
    return x + y + 2 * u64_build(__umulhi(xlo, ylo), xlo * ylo);
}

template<uint32_t bw, uint32_t bh, uint32_t dx, uint32_t dy, uint32_t offset>
__device__ void g(struct block_l *block, uint32_t subblock, uint32_t hash_lane)
{
    uint32_t index[4];
    for (uint32_t i = 0; i < 4; i++) {
        uint32_t bpos = (hash_lane + i * offset) % 4;
        uint32_t x = (subblock * dy + i * dx) * bw + bpos % bw;
        uint32_t y = (subblock * dx + i * dy) * bh + bpos / bw;

        index[i] = y * 16 + (x + (y / 2) * 4) % 16;
    }

    uint64_t a, b, c, d;
    a = u64_build(block->hi[index[0]], block->lo[index[0]]);
    b = u64_build(block->hi[index[1]], block->lo[index[1]]);
    c = u64_build(block->hi[index[2]], block->lo[index[2]]);
    d = u64_build(block->hi[index[3]], block->lo[index[3]]);

    a = f(a, b);
    d = rotr64(d ^ a, 32);
    c = f(c, d);
    b = rotr64(b ^ c, 24);
    a = f(a, b);
    d = rotr64(d ^ a, 16);
    c = f(c, d);
    b = rotr64(b ^ c, 63);

    block->lo[index[0]] = u64_lo(a);
    block->lo[index[1]] = u64_lo(b);
    block->lo[index[2]] = u64_lo(c);
    block->lo[index[3]] = u64_lo(d);

    block->hi[index[0]] = u64_hi(a);
    block->hi[index[1]] = u64_hi(b);
    block->hi[index[2]] = u64_hi(c);
    block->hi[index[3]] = u64_hi(d);
}

__device__ void shuffle_block(uint32_t thread, struct block_l *block)
{
    uint32_t subblock = (thread >> 2) & 0x7;
    uint32_t hash_lane = (thread >> 0) & 0x3;

    g<4, 1, 1, 0, 0>(block, subblock, hash_lane);

    __syncthreads();

    g<4, 1, 1, 0, 1>(block, subblock, hash_lane);

    __syncthreads();

    g<2, 2, 0, 1, 0>(block, subblock, hash_lane);

    __syncthreads();

    g<2, 2, 0, 1, 1>(block, subblock, hash_lane);
}

__device__ void next_addresses(uint32_t thread,
                               struct block_l *addr, struct block_l *tmp,
                               uint32_t thread_input)
{
    addr->lo[thread] = thread_input;
    addr->hi[thread] = 0;
    for (uint32_t i = 1; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos = i * THREADS_PER_LANE + thread;
        addr->hi[pos] = addr->lo[pos] = 0;
    }

    __syncthreads();

    shuffle_block(thread, addr);

    __syncthreads();

    addr->lo[thread] ^= thread_input;
    move_block(tmp, addr, thread);

    __syncthreads();

    shuffle_block(thread, addr);

    __syncthreads();

    xor_block(addr, tmp, thread);

    __syncthreads();
}

__device__ void compute_ref_pos(
        uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t lane, uint32_t slice, uint32_t offset,
        uint32_t *ref_lane, uint32_t *ref_index)
{
    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    *ref_lane = *ref_lane % lanes;

    uint32_t base;
    if (pass != 0) {
        base = lane_blocks - segment_blocks;
    } else {
        if (slice == 0) {
            *ref_lane = lane;
        }
        base = slice * segment_blocks;
    }

    uint32_t ref_area_size = base + offset - 1;
    if (*ref_lane != lane) {
        ref_area_size = min(ref_area_size, base);
    }

    *ref_index = __umulhi(*ref_index, *ref_index);
    *ref_index = ref_area_size - 1 - __umulhi(ref_area_size, *ref_index);

    if (pass != 0 && slice != ARGON2_SYNC_POINTS - 1) {
        *ref_index += (slice + 1) * segment_blocks;
        if (*ref_index >= lane_blocks) {
            *ref_index -= lane_blocks;
        }
    }
}

struct ref {
    uint32_t ref_lane;
    uint32_t ref_index;
};

struct shmem_precompute {
    struct block_l addr, tmp;
};

/*
 * Refs hierarchy:
 * lanes -> passes -> slices -> blocks
 */
__global__ void argon2i_precompute_kernel(
        struct ref *refs, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    uint32_t block_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t warp = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t segment_addr_blocks = (segment_blocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t block = block_id % segment_addr_blocks;
    uint32_t segment = block_id / segment_addr_blocks;

    uint32_t slice = segment % ARGON2_SYNC_POINTS;
    uint32_t pass_id = segment / ARGON2_SYNC_POINTS;

    uint32_t pass = pass_id % passes;
    uint32_t lane = pass_id / passes;

    extern __shared__ struct shmem_precompute shared_mem2[];

    struct block_l *addr = &shared_mem2[warp].addr;
    struct block_l *tmp = &shared_mem2[warp].tmp;

    uint32_t thread_input;
    switch (thread) {
    case 0:
        thread_input = pass;
        break;
    case 1:
        thread_input = lane;
        break;
    case 2:
        thread_input = slice;
        break;
    case 3:
        thread_input = lanes * segment_blocks * ARGON2_SYNC_POINTS;
        break;
    case 4:
        thread_input = passes;
        break;
    case 5:
        thread_input = ARGON2_I;
        break;
    case 6:
        thread_input = block + 1;
        break;
    default:
        thread_input = 0;
        break;
    }

    next_addresses(thread, addr, tmp, thread_input);

    refs += segment * segment_blocks;

    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos_l = (thread & 0x10) + ((thread + i * 4) & 0xf);
        uint32_t ref_index = addr->lo[i * THREADS_PER_LANE + pos_l];
        uint32_t ref_lane  = addr->hi[i * THREADS_PER_LANE + pos_l];

        uint32_t pos = i * THREADS_PER_LANE + thread;
        uint32_t offset = block * ARGON2_QWORDS_IN_BLOCK + pos;
        if (offset < segment_blocks) {
            compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                            &ref_lane, &ref_index);

            refs[offset].ref_index = ref_index;
            refs[offset].ref_lane  = ref_lane;
        }
    }
}

template<uint32_t version>
__device__ void argon2_core(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_l *prev, struct block_l *tmp,
        uint32_t lane_blocks, uint32_t thread, uint32_t pass,
        uint32_t ref_index, uint32_t ref_lane)
{
    struct block_g *mem_ref = memory + ref_lane * lane_blocks + ref_index;

    if (version != ARGON2_VERSION_10 && pass != 0) {
        load_block(tmp, mem_curr, thread);
        load_block_xor(prev, mem_ref, thread);
        xor_block(tmp, prev, thread);
    } else {
        load_block_xor(prev, mem_ref, thread);
        move_block(tmp, prev, thread);
    }

    __syncthreads();

    shuffle_block(thread, prev);

    __syncthreads();

    xor_block(prev, tmp, thread);

    store_block(mem_curr, prev, thread);
}

template<uint32_t version>
__global__ void argon2i_kernel_segment_precompute(
        struct block_g *memory, const struct ref *refs,
        uint32_t passes, uint32_t lanes, uint32_t segment_blocks,
        uint32_t pass, uint32_t slice)
{
    extern __shared__ struct block_l shared_mem[];
    struct block_l *shared = shared_mem;

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += job_id * lanes * lane_blocks;
    /* select warp's shared memory buffer: */
    shared += threadIdx.y * 2;

    struct block_l *prev = &shared[0];
    struct block_l *tmp  = &shared[1];

    struct block_g *mem_segment =
            memory + lane * lane_blocks + slice * segment_blocks;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1;
            mem_curr = mem_segment + 2;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - 1;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks : 0) - 1;
        mem_curr = mem_segment;
    }

    load_block(prev, mem_prev, thread);

    refs += (lane * passes + pass) * lane_blocks + slice * segment_blocks;
    refs += start_offset;

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_core<version>(memory, mem_curr, prev, tmp, lane_blocks,
                             thread, pass, refs->ref_index, refs->ref_lane);

        ++mem_curr;
        ++refs;
    }
}

template<uint32_t version>
__global__ void argon2i_kernel_oneshot_precompute(
        struct block_g *memory, const struct ref *refs, uint32_t passes,
        uint32_t lanes, uint32_t segment_blocks)
{
    extern __shared__ struct block_l shared_mem[];
    struct block_l *shared = shared_mem;

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += job_id * lanes * lane_blocks;
    /* select lane's shared memory buffer: */
    shared += lane * 2;

    struct block_l *prev = &shared[0];
    struct block_l *tmp  = &shared[1];

    struct block_g *mem_lane = memory + lane * lane_blocks;
    struct block_g *mem_prev = mem_lane + 1;
    struct block_g *mem_curr = mem_lane + 2;

    load_block(prev, mem_prev, thread);

    refs += lane * passes * lane_blocks + 2;

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_core<version>(memory, mem_curr, prev, tmp,
                                     lane_blocks, thread, pass,
                                     refs->ref_index, refs->ref_lane);

                ++mem_curr;
                ++refs;
            }

            __syncthreads();
        }

        mem_curr = mem_lane;
    }
}

template<uint32_t type, uint32_t version>
__device__ void argon2_step(
        struct block_g *memory, struct block_g *mem_curr,
        struct block_l *prev, struct block_l *tmp, struct block_l *addr,
        uint32_t lanes, uint32_t segment_blocks, uint32_t lane_blocks,
        uint32_t thread, uint32_t *thread_input,
        uint32_t lane, uint32_t pass, uint32_t slice, uint32_t offset)
{
    uint32_t ref_index, ref_lane;

    if (type == ARGON2_I) {
        uint32_t addr_index = offset % ARGON2_QWORDS_IN_BLOCK;
        if (addr_index == 0) {
            if (thread == 6) {
                ++*thread_input;
            }
            next_addresses(thread, addr, tmp, *thread_input);
        }
        uint32_t addr_index_x = addr_index % 16;
        uint32_t addr_index_y = addr_index / 16;
        addr_index = addr_index_y * 16 +
                (addr_index_x + (addr_index_y / 2) * 4) % 16;
        ref_index = addr->lo[addr_index];
        ref_lane = addr->hi[addr_index];
    } else {
        ref_index = prev->lo[0];
        ref_lane = prev->hi[0];
    }

    compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                    &ref_lane, &ref_index);

    argon2_core<version>(memory, mem_curr, prev, tmp, lane_blocks,
                         thread, pass, ref_index, ref_lane);
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks, uint32_t pass, uint32_t slice)
{
    extern __shared__ struct block_l shared_mem[];
    struct block_l *shared = shared_mem;

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += job_id * lanes * lane_blocks;
    /* select warp's shared memory buffer: */
    shared += threadIdx.y * (type == ARGON2_I ? 3 : 2);

    uint32_t thread_input;
    struct block_l *prev = &shared[0];
    struct block_l *tmp  = &shared[1];
    struct block_l *addr;

    if (type == ARGON2_I) {
        addr = &shared[2];

        switch (thread) {
        case 0:
            thread_input = pass;
            break;
        case 1:
            thread_input = lane;
            break;
        case 2:
            thread_input = slice;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = ARGON2_I;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (pass == 0 && slice == 0 && segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(thread, addr, tmp, thread_input);
        }
    }

    struct block_g *mem_segment =
            memory + lane * lane_blocks + slice * segment_blocks;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1;
            mem_curr = mem_segment + 2;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - 1;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks : 0) - 1;
        mem_curr = mem_segment;
    }

    load_block(prev, mem_prev, thread);

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step<type, version>(
                    memory, mem_curr, prev, tmp, addr,
                    lanes, segment_blocks, lane_blocks,
                    thread, &thread_input,
                    lane, pass, slice, offset);

        ++mem_curr;
    }
}

template<uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot(
        struct block_g *memory, uint32_t passes, uint32_t lanes,
        uint32_t segment_blocks)
{
    extern __shared__ struct block_l shared_mem[];
    struct block_l *shared = shared_mem;

    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane   = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += job_id * lanes * lane_blocks;
    /* select lane's shared memory buffer: */
    shared += lane * (type == ARGON2_I ? 3 : 2);

    struct block_l *prev = &shared[0];
    struct block_l *tmp  = &shared[1];
    struct block_l *addr;
    uint32_t thread_input;

    if (type == ARGON2_I) {
        addr = &shared[2];

        switch (thread) {
        case 1:
            thread_input = lane;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = ARGON2_I;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(thread, addr, tmp, thread_input);
        }
    }

    struct block_g *mem_lane = memory + lane * lane_blocks;
    struct block_g *mem_prev = mem_lane + 1;
    struct block_g *mem_curr = mem_lane + 2;

    load_block(prev, mem_prev, thread);

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step<type, version>(
                            memory, mem_curr, prev, tmp, addr,
                            lanes, segment_blocks, lane_blocks,
                            thread, &thread_input,
                            lane, pass, slice, offset);

                ++mem_curr;
            }

            __syncthreads();

            if (type == ARGON2_I) {
                if (thread == 2) {
                    ++thread_input;
                }
                if (thread == 6) {
                    thread_input = 0;
                }
            }
        }
        if (type == ARGON2_I) {
            if (thread == 0) {
                ++thread_input;
            }
            if (thread == 2) {
                thread_input = 0;
            }
        }
        mem_curr = mem_lane;
    }
}

Argon2KernelRunner::Argon2KernelRunner(
        uint32_t type, uint32_t version, uint32_t passes, uint32_t lanes,
        uint32_t segmentBlocks, uint32_t batchSize, bool bySegment,
        bool precompute)
    : type(type), version(version), passes(passes), lanes(lanes),
      segmentBlocks(segmentBlocks), batchSize(batchSize), bySegment(bySegment),
      precompute(precompute), stream(nullptr), memory(nullptr),
      refs(nullptr), start(nullptr), end(nullptr)
{
    // FIXME: check overflow:
    uint32_t memorySize = lanes * segmentBlocks * ARGON2_SYNC_POINTS
            * ARGON2_BLOCK_SIZE * batchSize;

    CudaException::check(hipMallocManaged(&memory, memorySize,
                                           hipMemAttachHost));

    CudaException::check(hipEventCreate(&start));
    CudaException::check(hipEventCreate(&end));

    CudaException::check(hipStreamCreate(&stream));
    CudaException::check(hipStreamAttachMemAsync(stream, memory));
    CudaException::check(hipStreamSynchronize(stream));

    if (type == ARGON2_I && precompute) {
        uint32_t segments = passes * lanes * ARGON2_SYNC_POINTS;

        uint32_t refsSize = segments * segmentBlocks * sizeof(struct ref);

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << refsSize << " bytes for refs..."
                  << std::endl;
#endif

        CudaException::check(hipMallocManaged(&refs, refsSize,
                                               hipMemAttachHost));

        CudaException::check(hipStreamAttachMemAsync(stream, refs));
        CudaException::check(hipStreamSynchronize(stream));

        precomputeRefs();
        CudaException::check(hipStreamSynchronize(stream));
    }
}

void Argon2KernelRunner::precomputeRefs()
{
    struct ref *refs = (struct ref *)this->refs;

    uint32_t segmentAddrBlocks = (segmentBlocks + ARGON2_QWORDS_IN_BLOCK - 1)
            / ARGON2_QWORDS_IN_BLOCK;
    uint32_t segments = passes * lanes * ARGON2_SYNC_POINTS;

    dim3 blocks = dim3(1, segments * segmentAddrBlocks);
    dim3 threads = dim3(THREADS_PER_LANE);

    uint32_t shmemSize = sizeof(struct shmem_precompute);
    argon2i_precompute_kernel<<<blocks, threads, shmemSize, stream>>>(
            refs, passes, lanes, segmentBlocks);
}

Argon2KernelRunner::~Argon2KernelRunner()
{
    if (start != nullptr) {
        hipEventDestroy(start);
    }
    if (end != nullptr) {
        hipEventDestroy(end);
    }
    if (stream != nullptr) {
        hipStreamDestroy(stream);
    }
    if (memory != nullptr) {
        hipFree(memory);
    }
    if (refs != nullptr) {
        hipFree(refs);
    }
}

void Argon2KernelRunner::runKernelSegment(uint32_t lanesPerBlock,
                                          uint32_t jobsPerBlock,
                                          uint32_t pass, uint32_t slice)
{
    if (lanesPerBlock > lanes || lanes % lanesPerBlock != 0) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, lanes / lanesPerBlock, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanesPerBlock, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    if (type == ARGON2_I) {
        if (precompute) {
            uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 2;
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2i_kernel_segment_precompute<ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2i_kernel_segment_precompute<ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        } else {
            uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 3;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks,
                            pass, slice);
            }
        }
    } else {
        uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 2;
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        } else {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
        }
    }
}

void Argon2KernelRunner::runKernelOneshot(uint32_t lanesPerBlock,
                                          uint32_t jobsPerBlock)
{
    if (lanesPerBlock != lanes) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g *memory_blocks = (struct block_g *)memory;
    dim3 blocks = dim3(1, 1, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanes, jobsPerBlock);
    uint32_t blockSize = lanesPerBlock * jobsPerBlock;
    if (type == ARGON2_I) {
        if (precompute) {
            uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 2;
            struct ref *refs = (struct ref *)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2i_kernel_oneshot_precompute<ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2i_kernel_oneshot_precompute<ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 3;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_10>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_13>
                        <<<blocks, threads, shared_size, stream>>>(
                            memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else {
        uint32_t shared_size = blockSize * ARGON2_BLOCK_SIZE * 2;
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_10>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        } else {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_13>
                    <<<blocks, threads, shared_size, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
        }
    }
}

void Argon2KernelRunner::run(uint32_t lanesPerBlock, uint32_t jobsPerBlock)
{
    CudaException::check(hipEventRecord(start, stream));

    if (bySegment) {
        for (uint32_t pass = 0; pass < passes; pass++) {
            for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; slice++) {
                runKernelSegment(lanesPerBlock, jobsPerBlock, pass, slice);
            }
        }
    } else {
        runKernelOneshot(lanesPerBlock, jobsPerBlock);
    }

    CudaException::check(hipGetLastError());

    CudaException::check(hipEventRecord(end, stream));
}

float Argon2KernelRunner::finish()
{
    CudaException::check(hipStreamSynchronize(stream));

    float time = 0.0;
    CudaException::check(hipEventElapsedTime(&time, start, end));
    return time;
}

} // cuda
} // argon2
